#include "hip/hip_runtime.h"
#include "filter.h"

__global__ void helloFromGPU()
{
    printf("Hello from GPU, threadId %d!\n", threadIdx.x);
    printf("Goodbye from GPU, threadId %d!\n", threadIdx.x);
}

void invoke_kernel()
{
  helloFromGPU<<<1, 64>>>(); // 1 group of 64 threads do this function in parallel
  hipDeviceReset(); // Force to print
}