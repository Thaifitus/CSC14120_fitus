#include "hip/hip_runtime.h"
#ifndef USE_CUDA

#include "filter.h"

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess)                                \
    {                                                        \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(EXIT_FAILURE);                                    \
    }                                                        \
  }

__global__ void filter1(float *d_in, int channel_in, int height_in, int width_in,
                        float *d_out,
                        float *filter, int filterWidth, int n)
{
  // Check images input
  // int d_i = 0;
  // for (int col = 0; col < n; ++col)
  // {
  //   for (int row = 0; row < channel_in * height_in * width_in; ++row)
  //   {
  //     printf("%f ", d_in[d_i++]);
  //   }
  //   printf("\n");
  // }


  return;
}

int invoke_kernel(const float *h_in, int channel_in, int height_in, int width_in,
                  float *&h_out, int height_out, int width_out, int channel_out,
                  int n_sample, int filter_type,
                  float *filter, int filterWidth, int stride, int pad_w, int pad_h)
{
  // TODO: Allocate device memories
  float *d_in, *d_out, *d_filter;
  size_t nBytes_d_in = height_in * width_in * channel_in * n_sample * sizeof(float);
  size_t nBytes_d_out = height_out * width_out * channel_out * n_sample * sizeof(float);
  size_t nBytes_d_filter = channel_in * filterWidth * filterWidth * channel_out * sizeof(float);

  CHECK(hipMalloc((void **)&d_in, nBytes_d_in));
  CHECK(hipMalloc((void **)&d_out, nBytes_d_out));
  CHECK(hipMalloc((void **)&d_filter, nBytes_d_filter));

  // TODO: Copy data to device memories
  CHECK(hipMemcpy(d_in, h_in, nBytes_d_in, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_filter, filter, nBytes_d_filter, hipMemcpyHostToDevice));

  // TODO: Set grid size and call kernel
  dim3 gridSize(1);
  dim3 blockSize(32);
  if (filter_type == 1)
  {
    filter1<<<gridSize, blockSize>>>(d_in, channel_in, height_in, width_in,
                                     d_out,
                                     d_filter, filterWidth, n_sample);
    // Checks for synchronous errors
    hipError_t errSync = hipGetLastError();
    if (errSync != hipSuccess)
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  }

  // TODO: Copy result from device memory
  // CHECK(hipMemcpy(h_out, d_out, nBytes_d_out, hipMemcpyDeviceToHost));

  // TODO: Free device memories
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_out));
  CHECK(hipFree(d_filter));

  // hipDeviceReset(); // Force to print

  // return filter type
  return 0;
}

#endif